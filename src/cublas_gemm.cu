hipblasHandle_t handle;

void cublas_sgemm(const float *A, const float *B, float *C, int M, int N, int K)
{
    float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T,
                M, N, K,
                /*alpha=*/&alpha,
                A, M,
                B, N,
                /*beta=*/&beta,
                C, M);
}