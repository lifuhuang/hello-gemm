#include "cublas_sgemm.hpp"
#include <hipblas.h>

void cublas_sgemm(hipblasHandle_t handle, const float *A, const float *B, float *C, int M, int N, int K)
{
    float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T,
                M, N, K,
                /*alpha=*/&alpha,
                A, M,
                B, N,
                /*beta=*/&beta,
                C, M);
}